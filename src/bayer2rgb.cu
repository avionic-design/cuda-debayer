#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2016 Avionic Design GmbH
 * Meike Vocke <meike.vocke@avionic-design.de>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License version 2 as
 * published by the Free Software Foundation.
 *
 * This file contains the CUDA kernel with functions to initialise all needed
 * parameters for kernel Launch. Also the destruction of generated parameter is
 * included.
 *
 * Compute Capability 3.0 or higher required
 */

#include "bayer2rgb.h"

#define PIX(x, y, imgw)		((x) + (y) * (imgw))
#define LEFT(x, y, imgw)	((x) - 1 + (y) * (imgw))
#define RIGHT(x, y, imgw)	((x) + 1 + (y) * (imgw))
#define TOP(x, y, imgw)		((x) + ((y) - 1) * (imgw))
#define BOT(x, y, imgw)		((x) + ((y) + 1) * (imgw))
#define TL(x, y, imgw)		((x) - 1 + ((y) - 1) * (imgw))
#define BL(x, y, imgw)		((x) - 1 + ((y) + 1) * (imgw))
#define TR(x, y, imgw)		((x) + 1 + ((y) - 1) * (imgw))
#define BR(x, y, imgw)		((x) + 1 + ((y) + 1) * (imgw))

struct cuda_vars {
	hipArray *data[2];

	dim3 threads_p_block;
	dim3 blocks_p_grid;

	uint8_t *d_bilinear[2];
	uint8_t *d_input[2];

	uint32_t width;
	uint32_t height;

	hipStream_t streams[2];

	uint8_t cnt;
	uint8_t bpp;
};

/**
 * CUDA Kernel Device code for RGGB
 *
 * Computes the Bilear Interpolation of missing coloured pixel from Bayer pattern.
 * Output is RGB.
 */
__global__ void bayer_to_rgb(uint8_t *in, uint8_t *out, uint32_t imgw,
		uint32_t imgh, uint8_t bpp)
{
	int x = 2 * ((blockDim.x * blockIdx.x) + threadIdx.x);
	int y = 2 * ((blockDim.y * blockIdx.y) + threadIdx.y);
	int elemCols = imgw * bpp;

	if ((x + 2) < imgw && (x - 1) >= 0 && (y + 2) < imgh && (y - 1) >= 0) {
		/* red at red */
		out[y * elemCols + x * bpp] = in[PIX(x, y, imgw)];
		/* green at red */
		out[y * elemCols + x * bpp + 1] =
				((uint32_t)in[TOP(x, y, imgw)] +
				in[BOT(x, y, imgw)] +
				in[LEFT(x, y, imgw)] +
				in[RIGHT(x, y, imgw)]) / 4;
		/* blue at red */
		out[y * elemCols + x * bpp + 2] =
				((uint32_t)in[TL(x, y, imgw)] +
				in[TR(x, y, imgw)] +
				in[BL(x, y, imgw)] +
				in[BR(x, y, imgw)]) / 4;

		/* red at upper right green */
		out[y * elemCols + (x + 1) * bpp] =
				((uint32_t)in[LEFT(x + 1, y, imgw)] +
				in[RIGHT(x + 1, y, imgw)]) / 2;
		/* green at upper right green */
		out[y * elemCols + (x + 1) * bpp + 1] =
				in[PIX(x + 1, y, imgw)];
		/* blue at upper right green */
		out[y * elemCols + (x + 1) * bpp + 2] =
				((uint32_t)in[TOP(x + 1, y, imgw)] +
				in[BOT(x + 1, y, imgw)]) / 2;

		/* red at lower left green */
		out[(y + 1) * elemCols + x * bpp] =
				((uint32_t)in[TOP(x, y + 1, imgw)] +
				in[BOT(x, y + 1, imgw)]) / 2;
		/* green at lower left green */
		out[(y + 1) * elemCols + x * bpp + 1] =
				in[PIX(x, y + 1, imgw)];
		/* blue at lower left green */
		out[(y + 1) * elemCols + x * bpp + 2] =
				((uint32_t)in[LEFT(x, y + 1, imgw)] +
				in[RIGHT(x, y + 1, imgw)]) / 2;

		/* red at blue */
		out[(y + 1) * elemCols + (x + 1) * bpp] =
				((uint32_t)in[TL(x + 1, y + 1, imgw)] +
				in[TR(x + 1, y + 1, imgw)] +
				in[BL(x + 1, y + 1, imgw)] +
				in[BR(x + 1, y + 1, imgw)]) / 4;
		/* green at blue */
		out[(y + 1) * elemCols + (x + 1) * bpp + 1] =
				((uint32_t)in[TOP(x + 1, y + 1, imgw)] +
				in[BOT(x + 1, y + 1, imgw)] +
				in[LEFT(x + 1, y + 1, imgw)] +
				in[RIGHT(x + 1, y + 1, imgw)]) / 4;
		/* blue at blue */
		out[(y + 1) * elemCols + (x + 1) * bpp + 2] =
				in[PIX(x + 1, y + 1, imgw)];

		if (bpp == 4) {
			out[y * elemCols + x * bpp + 3] = 255;
			out[y * elemCols + (x + 1) * bpp + 3] = 255;
			out[(y + 1) * elemCols + x * bpp + 3] = 255;
			out[(y + 1) * elemCols + (x + 1) * bpp + 3] = 255;
		}
	}
}

hipError_t bayer2rgb_process(struct cuda_vars *gpu_vars, const void *p,
		uint8_t **output, hipStream_t *stream, bool get_dev_ptr)
{
	hipError_t ret_val;

	if (gpu_vars == NULL)
		return hipErrorNotInitialized;

	ret_val = hipMemcpyAsync(gpu_vars->d_input[(gpu_vars->cnt % 2)],
			p, gpu_vars->width * gpu_vars->height *
			sizeof(uint8_t), hipMemcpyHostToDevice,
			gpu_vars->streams[gpu_vars->cnt % 2]);
	if (ret_val != hipSuccess) {
		fprintf(stderr, "Host to Device %d, %s\n", gpu_vars->cnt % 2,
				hipGetErrorString(ret_val));
		return ret_val;
	}

	bayer_to_rgb<<<gpu_vars->blocks_p_grid,
			gpu_vars->threads_p_block, 0,
			gpu_vars->streams[(gpu_vars->cnt % 2)]
		>>>(gpu_vars->d_input[(gpu_vars->cnt % 2)],
			gpu_vars->d_bilinear[(gpu_vars->cnt % 2)],
			gpu_vars->width, gpu_vars->height, gpu_vars->bpp);

	if (get_dev_ptr) {
		*output = (uint8_t *)gpu_vars->d_bilinear[(gpu_vars->cnt % 2)];
	} else {
		ret_val = hipMemcpyAsync(*output,
			gpu_vars->d_bilinear[gpu_vars->cnt % 2],
			gpu_vars->width * gpu_vars->height * sizeof(uint8_t) *
			gpu_vars->bpp, hipMemcpyDeviceToHost,
			gpu_vars->streams[gpu_vars->cnt % 2]);
		if (ret_val != hipSuccess) {
			fprintf(stderr, "Device to Host %d, %s\n",
					gpu_vars->cnt % 2,
					hipGetErrorString(ret_val));
			return ret_val;
		}
	}

	ret_val = hipStreamSynchronize(gpu_vars->streams[
			gpu_vars->cnt % 2]);
	if (ret_val != hipSuccess) {
		fprintf(stderr, "device synchronize\n");
		return ret_val;
	}

	gpu_vars->cnt = (gpu_vars->cnt + 1) % 2;

	return hipSuccess;
}

hipError_t alloc_create_cuda_data(struct cuda_vars *gpu_vars, uint8_t cnt)
{
	hipError_t ret_val = hipSuccess;

	ret_val = hipMalloc(&gpu_vars->d_input[cnt], gpu_vars->width *
			gpu_vars->height * sizeof(uint8_t));
	if (ret_val != hipSuccess) {
		fprintf(stderr, "hipMalloc d_bilinear %d, %s\n", cnt,
				hipGetErrorString(ret_val));
		return ret_val;
	}

	ret_val = hipMalloc(&gpu_vars->d_bilinear[cnt], gpu_vars->width *
			gpu_vars->height * sizeof(uint8_t) *
			gpu_vars->bpp);
	if (ret_val != hipSuccess) {
		fprintf(stderr, "hipMalloc d_bilinear %d, %s\n", cnt,
				hipGetErrorString(ret_val));
		return ret_val;
	}

	ret_val = hipStreamCreate(&gpu_vars->streams[cnt]);
	if (ret_val != hipSuccess) {
		fprintf(stderr, "hipStreamCreate %d, %s\n", cnt,
				hipGetErrorString(ret_val));
		return ret_val;
	}

	return ret_val;
}

hipError_t bayer2rgb_init(struct cuda_vars **gpu_vars_p, uint32_t width,
		uint32_t height, uint8_t bpp)
{
	struct cuda_vars *gpu_vars;
	hipError_t ret_val;
	int i;

	if (gpu_vars_p == NULL)
		return hipErrorNotInitialized;

	gpu_vars = (cuda_vars *) new(struct cuda_vars);
	if (!gpu_vars)
		return hipErrorOutOfMemory;

	gpu_vars->width = width;
	gpu_vars->height = height;
	gpu_vars->cnt = 0;
	gpu_vars->bpp = bpp;

	for (i = 0; i < 2; i++) {
		ret_val = alloc_create_cuda_data(gpu_vars, i);
		if (ret_val != hipSuccess)
			goto cleanup;
	}

	gpu_vars->threads_p_block = dim3(32, 32);
	gpu_vars->blocks_p_grid.x = (gpu_vars->width / 2 +
			gpu_vars->threads_p_block.x - 1) /
			gpu_vars->threads_p_block.x;
	gpu_vars->blocks_p_grid.y = (gpu_vars->height / 2 +
			gpu_vars->threads_p_block.y - 1) /
			gpu_vars->threads_p_block.y;

	*gpu_vars_p = gpu_vars;

	return hipSuccess;

cleanup:
	bayer2rgb_free(gpu_vars);

	return ret_val;
}

void free_cuda_data(struct cuda_vars *gpu_vars, uint8_t cnt)
{
	if (gpu_vars->d_input[cnt])
		hipFree(gpu_vars->d_input[cnt]);
	if (gpu_vars->d_bilinear[cnt])
		hipFree(gpu_vars->d_bilinear[cnt]);
	hipStreamDestroy(gpu_vars->streams[cnt]);
}

hipError_t bayer2rgb_free(struct cuda_vars *gpu_vars)
{
	int i;

	for (i = 0; i < 2; i++) {
		free_cuda_data(gpu_vars, i);
	}

	free(gpu_vars);

	return hipSuccess;
}
