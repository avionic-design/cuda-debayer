#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2016 Avionic Design GmbH
 * Meike Vocke <meike.vocke@avionic-design.de>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License version 2 as
 * published by the Free Software Foundation.
 *
 * This file contains the CUDA kernel with functions to initialise all needed
 * parameters for kernel Launch. Also the destruction of generated parameter is
 * included.
 *
 * Compute Capability 3.0 or higher required
 */

#include <linux/videodev2.h>

#include "bayer2rgb.h"
#include "bayer2rgb_kernel.h"

#define LEFT(x, y, imgw)	((x) - 1 + (y) * (imgw))
#define RIGHT(x, y, imgw)	((x) + 1 + (y) * (imgw))
#define TOP(x, y, imgw)		((x) + ((y) - 1) * (imgw))
#define BOT(x, y, imgw)		((x) + ((y) + 1) * (imgw))
#define TL(x, y, imgw)		((x) - 1 + ((y) - 1) * (imgw))
#define BL(x, y, imgw)		((x) - 1 + ((y) + 1) * (imgw))
#define TR(x, y, imgw)		((x) + 1 + ((y) - 1) * (imgw))
#define BR(x, y, imgw)		((x) + 1 + ((y) + 1) * (imgw))

#define PIX(in, x, y, imgw) \
	in[((x) + (y) * (imgw))]

#define INTERPOLATE_H(in, x, y, w) \
	(((uint32_t)in[LEFT(x, y, w)] + in[RIGHT(x, y, w)]) / 2)

#define INTERPOLATE_V(in, x, y, w) \
	(((uint32_t)in[TOP(x, y, w)] + in[BOT(x, y, w)]) / 2)

#define INTERPOLATE_HV(in, x, y, w) \
	(((uint32_t)in[LEFT(x, y, w)] + in[RIGHT(x, y, w)] + \
		in[TOP(x, y, w)] + in[BOT(x, y, w)]) / 4)

#define INTERPOLATE_X(in, x, y, w) \
	(((uint32_t)in[TL(x, y, w)] + in[BL(x, y, w)] + \
		in[TR(x, y, w)] + in[BR(x, y, w)]) / 4)

#define RED 0
#define GREEN 1
#define BLUE 2

struct cuda_vars {
	hipArray *data[2];

	dim3 threads_p_block;
	dim3 blocks_p_grid;

	uint8_t *d_bilinear[2];
	uint8_t *d_input[2];

	uint32_t width;
	uint32_t height;

	hipStream_t streams[2];

	uint8_t cnt;
	uint8_t bpp;
};

/**
 * CUDA Kernel Device code for RGGB
 *
 * Computes the Bilear Interpolation of missing coloured pixel from Bayer pattern.
 * Output is RGB.
 *
 * Each CUDA thread computes four pixels in a 2x2 square. Therefore no if
 * conditions are required, which slows the CUDA kernels massively.
 *
 * The first square starts with the pixel in position 1,1. Therefore the square
 * for each thread looks like this:
 *
 * B G
 * G R
 *
 * This approach saves one pixel lines at the edges of the image in contrast to
 * the first square at 2,2 with:
 *
 * R G
 * G B
 *
 */
__global__ void bayer_to_rgb(uint8_t *in, uint8_t *out, uint32_t imgw,
		uint32_t imgh, uint8_t bpp)
{
	int x = 2 * ((blockDim.x * blockIdx.x) + threadIdx.x) + 1;
	int y = 2 * ((blockDim.y * blockIdx.y) + threadIdx.y) + 1;
	int elemCols = imgw * bpp;

	if ((x + 2) < imgw && (x - 1) >= 0 && (y + 2) < imgh && (y - 1) >= 0) {
		/* red at red */
		out[(y + 1) * elemCols + (x + 1) * bpp + RED] =
				PIX(in, x + 1, y + 1, imgw);
		/* green at red */
		out[(y + 1) * elemCols + (x + 1) * bpp + GREEN] =
				INTERPOLATE_HV(in, x + 1, y + 1, imgw);
		/* blue at red */
		out[(y + 1) * elemCols + (x + 1) * bpp + BLUE] =
				INTERPOLATE_X(in, x + 1, y + 1, imgw);

		/* red at lower left green */
		out[(y + 1) * elemCols + x * bpp + RED] =
				INTERPOLATE_H(in, x, y + 1, imgw);
		/* green at lower left green */
		out[(y + 1) * elemCols + x * bpp + GREEN] =
				PIX(in, x, y + 1, imgw);
		/* blue at lower left green */
		out[(y + 1) * elemCols + x * bpp + BLUE] =
				INTERPOLATE_V(in, x, y + 1, imgw);

		/* red at upper right green */
		out[y * elemCols + (x + 1) * bpp + RED] =
				INTERPOLATE_V(in, x + 1, y, imgw);
		/* green at upper right green */
		out[y * elemCols + (x + 1) * bpp + GREEN] =
				PIX(in, x + 1, y, imgw);
		/* blue at upper right green */
		out[y * elemCols + (x + 1) * bpp + BLUE] =
				INTERPOLATE_H(in, x + 1, y, imgw);

		/* red at blue */
		out[y * elemCols + x * bpp + RED] =
				INTERPOLATE_X(in, x, y, imgw);
		/* green at blue */
		out[y * elemCols + x * bpp + GREEN] =
				INTERPOLATE_HV(in, x, y, imgw);
		/* blue at blue */
		out[y * elemCols + x * bpp + BLUE] =
				PIX(in, x, y, imgw);

		if (bpp == 4) {
			out[y * elemCols + x * bpp + 3] = 255;
			out[y * elemCols + (x + 1) * bpp + 3] = 255;
			out[(y + 1) * elemCols + x * bpp + 3] = 255;
			out[(y + 1) * elemCols + (x + 1) * bpp + 3] = 255;
		}
	}
}

hipError_t bayer2rgb_process(struct cuda_vars *gpu_vars, const void *p,
		uint8_t **output, hipStream_t *stream, bool get_dev_ptr)
{
	hipError_t ret_val;

	if (gpu_vars == NULL)
		return hipErrorNotInitialized;

	ret_val = hipMemcpyAsync(gpu_vars->d_input[gpu_vars->cnt],
			p, gpu_vars->width * gpu_vars->height *
			sizeof(uint8_t), hipMemcpyHostToDevice,
			gpu_vars->streams[gpu_vars->cnt]);
	if (ret_val != hipSuccess) {
		fprintf(stderr, "Host to Device %d, %s\n", gpu_vars->cnt,
				hipGetErrorString(ret_val));
		return ret_val;
	}

	bayer_to_rgb<<<gpu_vars->blocks_p_grid,
			gpu_vars->threads_p_block, 0,
			gpu_vars->streams[gpu_vars->cnt]
		>>>(gpu_vars->d_input[gpu_vars->cnt],
			gpu_vars->d_bilinear[gpu_vars->cnt],
			gpu_vars->width, gpu_vars->height, gpu_vars->bpp);

	if (get_dev_ptr) {
		*output = (uint8_t *)gpu_vars->d_bilinear[gpu_vars->cnt];
	} else {
		ret_val = hipMemcpyAsync(*output,
			gpu_vars->d_bilinear[gpu_vars->cnt],
			gpu_vars->width * gpu_vars->height * sizeof(uint8_t) *
			gpu_vars->bpp, hipMemcpyDeviceToHost,
			gpu_vars->streams[gpu_vars->cnt]);
		if (ret_val != hipSuccess) {
			fprintf(stderr, "Device to Host %d, %s\n",
					gpu_vars->cnt,
					hipGetErrorString(ret_val));
			return ret_val;
		}
		ret_val = hipStreamSynchronize(
				gpu_vars->streams[gpu_vars->cnt]);
		if (ret_val != hipSuccess) {
			fprintf(stderr, "device synchronize\n");
			return ret_val;
		}
	}

	*stream = gpu_vars->streams[gpu_vars->cnt];

	gpu_vars->cnt = (gpu_vars->cnt + 1) % 2;

	return hipSuccess;
}

hipError_t alloc_create_cuda_data(struct cuda_vars *gpu_vars, uint8_t cnt)
{
	hipError_t ret_val = hipSuccess;

	ret_val = hipMalloc(&gpu_vars->d_input[cnt], gpu_vars->width *
			gpu_vars->height * sizeof(uint8_t));
	if (ret_val != hipSuccess) {
		fprintf(stderr, "hipMalloc d_bilinear %d, %s\n", cnt,
				hipGetErrorString(ret_val));
		return ret_val;
	}

	ret_val = hipMalloc(&gpu_vars->d_bilinear[cnt], gpu_vars->width *
			gpu_vars->height * sizeof(uint8_t) *
			gpu_vars->bpp);
	if (ret_val != hipSuccess) {
		fprintf(stderr, "hipMalloc d_bilinear %d, %s\n", cnt,
				hipGetErrorString(ret_val));
		return ret_val;
	}

	ret_val = hipStreamCreate(&gpu_vars->streams[cnt]);
	if (ret_val != hipSuccess) {
		fprintf(stderr, "hipStreamCreate %d, %s\n", cnt,
				hipGetErrorString(ret_val));
		return ret_val;
	}

	return ret_val;
}

hipError_t bayer2rgb_init(struct cuda_vars **gpu_vars_p, uint32_t width,
		uint32_t height, uint8_t bpp, uint32_t format)
{
	struct cuda_vars *gpu_vars;
	hipError_t ret_val;
	int i;

	if (gpu_vars_p == NULL)
		return hipErrorNotInitialized;

	gpu_vars = (cuda_vars *) new(struct cuda_vars);
	if (!gpu_vars)
		return hipErrorOutOfMemory;

	gpu_vars->width = width;
	gpu_vars->height = height;
	gpu_vars->cnt = 0;
	gpu_vars->bpp = bpp;

	switch (format) {
	case V4L2_PIX_FMT_SRGGB8:
		break;
	default:
		fprintf(stderr, "unsupported pixel format\n");
		ret_val = hipErrorInvalidValue;
		goto cleanup;
	}

	for (i = 0; i < 2; i++) {
		ret_val = alloc_create_cuda_data(gpu_vars, i);
		if (ret_val != hipSuccess)
			goto cleanup;
	}

	gpu_vars->threads_p_block = dim3(32, 32);
	gpu_vars->blocks_p_grid.x = (gpu_vars->width / 2 +
			gpu_vars->threads_p_block.x - 1) /
			gpu_vars->threads_p_block.x;
	gpu_vars->blocks_p_grid.y = (gpu_vars->height / 2 +
			gpu_vars->threads_p_block.y - 1) /
			gpu_vars->threads_p_block.y;

	*gpu_vars_p = gpu_vars;

	return hipSuccess;

cleanup:
	bayer2rgb_free(gpu_vars);

	return ret_val;
}

void free_cuda_data(struct cuda_vars *gpu_vars, uint8_t cnt)
{
	if (gpu_vars->d_input[cnt])
		hipFree(gpu_vars->d_input[cnt]);
	if (gpu_vars->d_bilinear[cnt])
		hipFree(gpu_vars->d_bilinear[cnt]);
	hipStreamDestroy(gpu_vars->streams[cnt]);
}

hipError_t bayer2rgb_free(struct cuda_vars *gpu_vars)
{
	int i;

	for (i = 0; i < 2; i++) {
		free_cuda_data(gpu_vars, i);
	}

	free(gpu_vars);

	return hipSuccess;
}
